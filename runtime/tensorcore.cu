
#include <hip/hip_runtime.h>
// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on
// Utility macro for this file

// MMA instruction wrappers:
//  The wrappers are subroutines that implement matrix of size
//    A(M,K) X B(K,N) = C(M,N)
//  The naming of the wrappers follow similar naming conventions
//    as the mma instructions.
//  All the mma macros follow the namespace and naming like
//    Arch::M (M-dim) N (N-dim) K(K-dim) (Layout), eg.
//    Volta::M16N16K4TT,
//  with the dimensions describing the size of the sub-matrices being
//   multiplied by this wrapper.
//  see [Operand Layout Convention] in mma_type.h for details on the layout
//   notation.

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 750))

namespace Turing {

__device__ inline void M16N8K16TN(
    Array<float, 4, 1>& C,
    Array<unsigned, 4, 1>& A,
    Array<unsigned, 2, 1>& B) {
  asm("mma.sync.aligned.m16n8k8.row.col.f32.f16.f16.f32 {%0,%1,%2,%3}, {%4,%5}, {%6}, {%7,%8,%9,%10};\n"
      : "=f"(C[0]), "=f"(C[1]), "=f"(C[2]), "=f"(C[3])
      : "r"(A[0]),
        "r"(A[1]),
        "r"(B[0]),
        "f"(C[0]),
        "f"(C[1]),
        "f"(C[2]),
        "f"(C[3]));
  asm("mma.sync.aligned.m16n8k8.row.col.f32.f16.f16.f32 {%0,%1,%2,%3}, {%4,%5}, {%6}, {%7,%8,%9,%10};\n"
      : "=f"(C[0]), "=f"(C[1]), "=f"(C[2]), "=f"(C[3])
      : "r"(A[2]),
        "r"(A[3]),
        "r"(B[1]),
        "f"(C[0]),
        "f"(C[1]),
        "f"(C[2]),
        "f"(C[3]));
}

__device__ inline void M16N16K16TN(
    Array<float, 8, 1>& C,
    Array<unsigned, 4, 1>& A,
    Array<unsigned, 4, 1>& B) {
  auto* _C = reinterpret_cast<Array<float, 4, 1>*>(&C);
  auto* _B = reinterpret_cast<Array<unsigned, 2, 1>*>(&B);
  M16N8K16TN(_C[0], A, _B[0]);
  M16N8K16TN(_C[1], A, _B[1]);
}

} // namespace Turing

#endif // Arch 75

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800))

namespace Ampere {

__device__ inline void M16N8K16TNF16(
    Array<float, 4, 1>& C,
    Array<unsigned, 4, 1>& A,
    Array<unsigned, 2, 1>& B) {
  asm("mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 {%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
      : "=f"(C[0]), "=f"(C[1]), "=f"(C[2]), "=f"(C[3])
      : "r"(A[0]),
        "r"(A[1]),
        "r"(A[2]),
        "r"(A[3]),
        "r"(B[0]),
        "r"(B[1]),
        "f"(C[0]),
        "f"(C[1]),
        "f"(C[2]),
        "f"(C[3]));
}

__device__ inline void M16N8K16TNBF16(
    Array<float, 4, 1>& C,
    Array<unsigned, 4, 1>& A,
    Array<unsigned, 2, 1>& B) {
  asm("mma.sync.aligned.m16n8k16.row.col.f32.bf16.bf16.f32 {%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
      : "=f"(C[0]), "=f"(C[1]), "=f"(C[2]), "=f"(C[3])
      : "r"(A[0]),
        "r"(A[1]),
        "r"(A[2]),
        "r"(A[3]),
        "r"(B[0]),
        "r"(B[1]),
        "f"(C[0]),
        "f"(C[1]),
        "f"(C[2]),
        "f"(C[3]));
}

__device__ inline void M16N16K16TNF16(
    Array<float, 8, 1>& C,
    Array<unsigned, 4, 1>& A,
    Array<unsigned, 4, 1>& B) {
  auto* _C = reinterpret_cast<Array<float, 4, 1>*>(&C);
  auto* _B = reinterpret_cast<Array<unsigned, 2, 1>*>(&B);
  M16N8K16TNF16(_C[0], A, _B[0]);
  M16N8K16TNF16(_C[1], A, _B[1]);
}

__device__ inline void M16N16K16TNBF16(
    Array<float, 8, 1>& C,
    Array<unsigned, 4, 1>& A,
    Array<unsigned, 4, 1>& B) {
  auto* _C = reinterpret_cast<Array<float, 4, 1>*>(&C);
  auto* _B = reinterpret_cast<Array<unsigned, 2, 1>*>(&B);
  M16N8K16TNBF16(_C[0], A, _B[0]);
  M16N8K16TNBF16(_C[1], A, _B[1]);
}

} // namespace Ampere

#endif // Arch 80
